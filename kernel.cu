#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <time.h>
#include <cstdlib>
#include <sstream>
#include <string>


#include "Data.h"

//  W, dW  in 3D

/*__device__ __host__ double W(double r, double h)
	{        
	const double Pi = 3.14159265;
	double k, c, tmp1;
	k = (double) fabs(r)/h;
	c= (double) 1/(Pi);
	if (k < 1.0) {tmp1 = (double) 1.0 - 1.5*k*k + 0.75 * k*k*k;}
	if ((k >= 1.0) && (k <= 2.0)) {tmp1 = (double) 0.25 * (2-k)*(2-k)*(2-k);}
	if (k > 2.0) {tmp1 = 0.0;}

	return  c/(h*h*h) * tmp1;
	}

__device__ __host__ double dW(double r, double h)
	{                
	const double Pi = 3.14159265;
	double k, c, tmp1;
	k = (double) r/h;
	c= (double) 1/(Pi);
	if (k < -2.0) {tmp1 = 0.0;}
	if ((k >= -2.0) && (k <= -1.0)) {tmp1 = (double) 0.75 * (2.0+k)*(2.0+k);}
	if ((k > -1.0) && (k < 0)) {tmp1 = (double) -3.0*k - 2.25 * k*k;}
	if ((k >= 0) && (k <= 1.0)) {tmp1 = (double) -3.0*k + 2.25 * k*k;}
	if ((k >= 1.0) && (k <= 2.0)) {tmp1 = (double) -0.75 * (2.0-k)*(2.0-k);}
	if (k > 2.0) {tmp1 = 0.0;}

	return  c/(h*h*h) * tmp1;
	}

*/

//  W,  dW  in 2D


__device__ __host__ double W(double r, double h)
	{
			const double Pi = 3.14159265;
	double k, c, tmp1;
	k = (double) fabs(r)/h;
	c= (double) 10/(7*Pi);
	if (k < 1.0) {tmp1 = (double) 1.0 - 1.5*k*k + 0.75 * k*k*k;}
	if ((k >= 1.0) && (k <= 2.0)) {tmp1 = (double) 0.25 * (2-k)*(2-k)*(2-k);}
	if (k > 2.0) {tmp1 = 0.0;}

	return  c/(h*h) * tmp1;
	}

__device__ __host__ double dW(double r, double h)
	{
			const double Pi = 3.14159265;
	double k, c, tmp1;
	k = (double) r/h;
	c= (double) 10/(7*Pi);
	if (k < -2.0) {tmp1 = 0.0;}
	if ((k >= -2.0) && (k <= -1.0)) {tmp1 = (double) 0.75 * (2.0+k)*(2.0+k);}
	if ((k > -1.0) && (k < 0)) {tmp1 = (double) -3.0*k - 2.25 * k*k;}
	if ((k >= 0) && (k <= 1.0)) {tmp1 = (double) -3.0*k + 2.25 * k*k;}
	if ((k >= 1.0) && (k <= 2.0)) {tmp1 = (double) -0.75 * (2.0-k)*(2.0-k);}
	if (k > 2.0) {tmp1 = 0.0;}

	return  c/(h*h*h) * tmp1;
	}




int main()
{
	char s[128];
	double dlh, h0, p_wall;
	double x_temp, z_temp;
	int i,j;

	in_file = fopen( "src/Init.txt", "r" );
	fgets(s, 128, in_file);
	fgets(s, 128, in_file);	sscanf(s, "%lf", &Xm);
	fgets(s, 128, in_file);	sscanf(s, "%lf", &Zm);
	fgets(s, 128, in_file);	sscanf(s, "%lf", &X_lq);
	fgets(s, 128, in_file);	sscanf(s, "%lf", &Z_lq);
	fgets(s, 128, in_file);	sscanf(s, "%d", &Ppm);
	fgets(s, 128, in_file); sscanf(s, "%lf", &tau);

	fclose(in_file);

	Pmax = 1000000;

	x = new double[Pmax];
	z = new double[Pmax];
	p = new double[Pmax];
	mas = new double[Pmax];
	rho = new double[Pmax];
	Vx = new double[Pmax];
	Vz = new double[Pmax];
	Ax = new double[Pmax];
	Az = new double[Pmax];
	h = new double[Pmax];
	Ind = new int[Pmax];
	Nn = new int[Pmax];

	dlh = (double) 1.0/Ppm;
	h0 = 4.3*dlh;
	p_wall = 8;

	mas_0 = rho0 / (Ppm);
	mas_0 = mas_0 / (Ppm);

	p_ind = -1;
	Im = int(Xm*Ppm);
	Jm = int(Zm*Ppm);


	// Liquid
	for (i = -Im; i <= Im; i++)
	for (j = 0; j <= Jm; j++)
		{
			x_temp = (double) i*dlh;
			z_temp = (double) j*dlh;
			if ((x_temp <= X_lq) && (z_temp <= Z_lq)){
			p_ind = p_ind + 1;
			x[p_ind] = x_temp;// + (rand()%100-50.0)/100000.0 * dlh;
			z[p_ind] = z_temp;// + (rand()%100-50.0)/100000.0 * dlh;
			rho[p_ind] = rho0;
			p[p_ind] = p0;
			mas[p_ind] = mas_0;
			Vx[p_ind] = 0;
			Vz[p_ind] = 0;
			Az[p_ind] = 0;
			Ind[p_ind] = 0;
			}
		}

	Pr=p_ind; // Last liquid particle

	// Left wall
	for (i = -Im-p_wall; i <= -Im-1; i++)
	for (j = -p_wall; j <= Jm; j++)
		{
			x_temp = (double) i*dlh;
			z_temp = (double) j*dlh;
			p_ind = p_ind + 1;
			x[p_ind] = x_temp;// + (rand()%100-50.0)/100000.0 * dlh;
			z[p_ind] = z_temp;// + (rand()%100-50.0)/100000.0 * dlh;
			rho[p_ind] = rho0;
			p[p_ind] = p0;
			mas[p_ind] = mas_0;
			Vx[p_ind] = 0;
			Vz[p_ind] = 0;
			Az[p_ind] = 0;
			Ind[p_ind] = 1;
		}

	// Right wall
	for (i = Im+1; i <= Im+p_wall; i++)
	for (j = -p_wall; j <= Jm; j++)
		{
			x_temp = (double) i*dlh;
			z_temp = (double) j*dlh;
			p_ind = p_ind + 1;
			x[p_ind] = x_temp;// + (rand()%100-50.0)/100000.0 * dlh;
			z[p_ind] = z_temp;// + (rand()%100-50.0)/100000.0 * dlh;
			rho[p_ind] = rho0;
			p[p_ind] = p0;
			mas[p_ind] = mas_0;
			Vx[p_ind] = 0;
			Vz[p_ind] = 0;
			Az[p_ind] = 0;
			Ind[p_ind] = 1;
		}

	// Bottom
	for (i = -Im; i <= Im; i++)
	for (j = -p_wall; j <= -1; j++)
		{
			x_temp = (double) i*dlh;
			z_temp = (double) j*dlh;
			p_ind = p_ind + 1;
			x[p_ind] = x_temp;// + (rand()%100-50.0)/100000.0 * dlh;
			z[p_ind] = z_temp;// + (rand()%100-50.0)/100000.0 * dlh;
			rho[p_ind] = rho0;
			p[p_ind] = p0;
			mas[p_ind] = mas_0;
			Vx[p_ind] = 0;
			Vz[p_ind] = 0;
			Az[p_ind] = 0;
			Ind[p_ind] = 1;
		}

	Pm=p_ind;




	printf ("Particle: %d \n", Pm);

	out_num = 0;

	out_num = out_num + 1;
	sprintf(out_name, "Data/%d.dat", out_num);
	out_file = fopen( out_name, "w" );
//	fprintf( cut_file, "t=%5.3f mks \n", Tm*1e6 );
	fprintf( out_file, "# x \t z \t rho \t P \t Vx \t Vz \t Ax \t Az \t Ind \t Nn \n" );

    for (p_ind=0; p_ind<=Pm; p_ind++)
    	{
    		fprintf(out_file, "%10.8lf\t%10.8lf\t%10.8lf\t%10.8lf\t%10.8lf\t%10.8lf\t%10.8lf\t%10.8lf\t%d\t%d \n",
    			                          x[p_ind], z[p_ind], rho[p_ind], p[p_ind]/p0, Vx[p_ind], Vz[p_ind], Ax[p_ind], Az[p_ind], Ind[p_ind], Nn[p_ind]);
		}

	fclose(out_file);



	printf ("The end \n");
	return 0;

}
