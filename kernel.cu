#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <time.h>
#include <cstdlib>
#include <sstream>
#include <string>


#include "Data.h"

//  W, dW  in 3D

/*__device__ __host__ double W(double r, double h)
	{        
	const double Pi = 3.14159265;
	double k, c, tmp1;
	k = (double) fabs(r)/h;
	c= (double) 1/(Pi);
	if (k < 1.0) {tmp1 = (double) 1.0 - 1.5*k*k + 0.75 * k*k*k;}
	if ((k >= 1.0) && (k <= 2.0)) {tmp1 = (double) 0.25 * (2-k)*(2-k)*(2-k);}
	if (k > 2.0) {tmp1 = 0.0;}

	return  c/(h*h*h) * tmp1;
	}

__device__ __host__ double dW(double r, double h)
	{                
	const double Pi = 3.14159265;
	double k, c, tmp1;
	k = (double) r/h;
	c= (double) 1/(Pi);
	if (k < -2.0) {tmp1 = 0.0;}
	if ((k >= -2.0) && (k <= -1.0)) {tmp1 = (double) 0.75 * (2.0+k)*(2.0+k);}
	if ((k > -1.0) && (k < 0)) {tmp1 = (double) -3.0*k - 2.25 * k*k;}
	if ((k >= 0) && (k <= 1.0)) {tmp1 = (double) -3.0*k + 2.25 * k*k;}
	if ((k >= 1.0) && (k <= 2.0)) {tmp1 = (double) -0.75 * (2.0-k)*(2.0-k);}
	if (k > 2.0) {tmp1 = 0.0;}

	return  c/(h*h*h) * tmp1;
	}

*/

//  W,  dW  in 2D


__device__ __host__ double W(double r, double h)
	{
			const double Pi = 3.14159265;
	double k, c, tmp1;
	k = (double) fabs(r)/h;
	c= (double) 10/(7*Pi);
	if (k < 1.0) {tmp1 = (double) 1.0 - 1.5*k*k + 0.75 * k*k*k;}
	if ((k >= 1.0) && (k <= 2.0)) {tmp1 = (double) 0.25 * (2-k)*(2-k)*(2-k);}
	if (k > 2.0) {tmp1 = 0.0;}

	return  c/(h*h) * tmp1;
	}

__device__ __host__ double dW(double r, double h)
	{
			const double Pi = 3.14159265;
	double k, c, tmp1;
	k = (double) r/h;
	c= (double) 10/(7*Pi);
	if (k < -2.0) {tmp1 = 0.0;}
	if ((k >= -2.0) && (k <= -1.0)) {tmp1 = (double) 0.75 * (2.0+k)*(2.0+k);}
	if ((k > -1.0) && (k < 0)) {tmp1 = (double) -3.0*k - 2.25 * k*k;}
	if ((k >= 0) && (k <= 1.0)) {tmp1 = (double) -3.0*k + 2.25 * k*k;}
	if ((k >= 1.0) && (k <= 2.0)) {tmp1 = (double) -0.75 * (2.0-k)*(2.0-k);}
	if (k > 2.0) {tmp1 = 0.0;}

	return  c/(h*h) * tmp1;
	}




int main()
{

	printf ("The end \n");
	return 0;

}
